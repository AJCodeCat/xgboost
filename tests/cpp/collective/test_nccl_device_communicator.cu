
#include <hip/hip_runtime.h>
/**
 * Copyright 2022-2023, XGBoost contributors
 */
#ifdef XGBOOST_USE_NCCL

#include <gtest/gtest.h>

#include <bitset>
#include <string>  // for string

#include "../../../src/collective/communicator-inl.cuh"
#include "../../../src/collective/nccl_device_communicator.cuh"
#include "../helpers.h"

namespace xgboost {
namespace collective {

TEST(NcclDeviceCommunicatorSimpleTest, ThrowOnInvalidDeviceOrdinal) {
  auto construct = []() { NcclDeviceCommunicator comm{-1}; };
  EXPECT_THROW(construct(), dmlc::Error);
}

TEST(NcclDeviceCommunicatorSimpleTest, SystemError) {
  try {
    dh::safe_nccl(ncclSystemError);
  } catch (dmlc::Error const& e) {
    auto str = std::string{e.what()};
    ASSERT_TRUE(str.find("environment variables") != std::string::npos);
  }
}

namespace {
void VerifyAllReduceBitwiseAND() {
  auto const rank = collective::GetRank();
  std::bitset<64> original{};
  original[rank] = true;
  HostDeviceVector<uint64_t> buffer({original.to_ullong()}, rank);
  collective::AllReduce<collective::Operation::kBitwiseAND>(rank, buffer.DevicePointer(), 1);
  collective::Synchronize(rank);
  EXPECT_EQ(buffer.HostVector()[0], 0ULL);
}
}  // anonymous namespace

TEST(NcclDeviceCommunicator, MGPUAllReduceBitwiseAND) {
  auto const n_gpus = common::AllVisibleGPUs();
  if (n_gpus <= 1) {
    GTEST_SKIP() << "Skipping MGPUAllReduceBitwiseAND test with # GPUs = " << n_gpus;
  }
  RunWithInMemoryCommunicator(n_gpus, VerifyAllReduceBitwiseAND);
}

namespace {
void VerifyAllReduceBitwiseOR() {
  auto const world_size = collective::GetWorldSize();
  auto const rank = collective::GetRank();
  std::bitset<64> original{};
  original[rank] = true;
  HostDeviceVector<uint64_t> buffer({original.to_ullong()}, rank);
  collective::AllReduce<collective::Operation::kBitwiseOR>(rank, buffer.DevicePointer(), 1);
  collective::Synchronize(rank);
  EXPECT_EQ(buffer.HostVector()[0], (1ULL << world_size) - 1);
}
}  // anonymous namespace

TEST(NcclDeviceCommunicator, MGPUAllReduceBitwiseOR) {
  auto const n_gpus = common::AllVisibleGPUs();
  if (n_gpus <= 1) {
    GTEST_SKIP() << "Skipping MGPUAllReduceBitwiseOR test with # GPUs = " << n_gpus;
  }
  RunWithInMemoryCommunicator(n_gpus, VerifyAllReduceBitwiseOR);
}

namespace {
void VerifyAllReduceBitwiseXOR() {
  auto const world_size = collective::GetWorldSize();
  auto const rank = collective::GetRank();
  std::bitset<64> original{~0ULL};
  original[rank] = false;
  HostDeviceVector<uint64_t> buffer({original.to_ullong()}, rank);
  collective::AllReduce<collective::Operation::kBitwiseXOR>(rank, buffer.DevicePointer(), 1);
  collective::Synchronize(rank);
  EXPECT_EQ(buffer.HostVector()[0], (1ULL << world_size) - 1);
}
}  // anonymous namespace

TEST(NcclDeviceCommunicator, MGPUAllReduceBitwiseXOR) {
  auto const n_gpus = common::AllVisibleGPUs();
  if (n_gpus <= 1) {
    GTEST_SKIP() << "Skipping MGPUAllReduceBitwiseXOR test with # GPUs = " << n_gpus;
  }
  RunWithInMemoryCommunicator(n_gpus, VerifyAllReduceBitwiseXOR);
}

}  // namespace collective
}  // namespace xgboost

#endif  // XGBOOST_USE_NCCL
